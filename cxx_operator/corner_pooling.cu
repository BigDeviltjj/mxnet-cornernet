#include "hip/hip_runtime.h"

#include <vector>
#include "./corner_pooling-inl.h"
#include <mshadow/cuda/tensor_gpu-inl.cuh>
#include <mshadow/tensor.h>


namespace mxnet {
namespace op {

template<typename DType>
__global__ void CornerPoolingForwardTBKernel( const int count,
	const DType* in_data, const int batch, const int channel,  const int height, const int width, DType* out_data,
	int h_step, int h_start, int h_end) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x * gridDim.y){
    using mshadow::red::limits::MinValue;
//    const int index = threadIdx.x + blockDim.x * ( gridDim.x * blockIdx.y + blockIdx.x);
    const int b = index / (channel * width);
    const int c = index % (channel * width) / width;
    const int w = index % (channel * width) % width;
    DType max_val = MinValue<DType>();
    in_data += ( b * channel + c )* height * width + w;
    out_data += ( b * channel + c )* height * width + w;

    for (int h{h_start}; h != h_end; h += h_step) {
      const int index = h * width;
      max_val = max_val > in_data[index] ? max_val : in_data[index];
      out_data[index] = max_val;
    }	
  }
}


template<typename DType>
__global__ void CornerPoolingBackwardTBKernel( const int count,
	const DType* out_data, const int batch, const int channel,  const int height, const int width, const DType* out_grad, DType* in_grad,
	int h_step, int h_start, int h_end) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x * gridDim.y){
 //   const int index = threadIdx.x + blockDim.x * ( gridDim.x * blockIdx.y + blockIdx.x);
    const int b = index / (channel * width);
    const int c = index % (channel * width) / width;
    const int w = index % (channel * width) % width;

    out_data += ( b * channel + c )* height * width + w;
    out_grad += ( b * channel + c )* height * width + w;
    in_grad += ( b * channel + c )* height * width + w;

    int max_h_idx = h_start;
    for (int h{h_start}; h != h_end; h += h_step) {
      const int index = h * width;
      if (out_data[index] != out_data[max_h_idx]) {
        max_h_idx = index;
      }
      in_grad[max_h_idx] += out_grad[index];
    }	
  }
}


template<typename DType>
__global__ void CornerPoolingForwardLRKernel( const int count,
	const DType* in_data,const int batch, const int channel,  const int height, const int width, DType* out_data,
	int w_step, int w_start, int w_end) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x * gridDim.y){
    using mshadow::red::limits::MinValue;
//    const int index = threadIdx.x + blockDim.x * ( gridDim.x * blockIdx.y + blockIdx.x);
    const int b = index / (channel * width);
    const int c = index % (channel * width) / width;
    const int h = index % (channel * width) % width;
    DType max_val = MinValue<DType>();
    in_data += (( b * channel + c )* height + h)* width;
    out_data += (( b * channel + c )* height + h)* width;

    for (int w{w_start}; w != w_end; w += w_step) {
      const int index = w;
      max_val = max_val > in_data[index] ? max_val : in_data[index];
      out_data[index] = max_val;
    }	
  }
}


template<typename DType>
__global__ void CornerPoolingBackwardLRKernel( const int count,
	const DType* out_data, const int batch, const int channel,  const int height, const int width, const DType* out_grad, DType* in_grad,
	int w_step, int w_start, int w_end) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x * gridDim.y) {
    const int b = index / (channel * width);
    const int c = index % (channel * width) / width;
    const int h = index % (channel * width) % width;
  
    out_data += (( b * channel + c )* height + h)* width;
    out_grad += (( b * channel + c )* height + h)* width;
    in_grad += (( b * channel + c )* height + h)* width;
  
    int max_w_idx = w_start;
    for (int w{w_start}; w != w_end; w += w_step) {
      const int index = w;
      if (out_data[index] != out_data[max_w_idx]) {
        max_w_idx = index;
      }
      in_grad[max_w_idx] += out_grad[index];
    }	
  }
}



template<typename DType>
inline void corner_pool(mshadow::Stream<gpu>* s, const DType* in_data, const TShape& ishape,
                 const int corner_pooling_type, OpReqType req_type, DType* out_data) {
  using mshadow::red::limits::MinValue;
//const TShape& oshape = ishape;
  CHECK_EQ(req_type, kWriteTo) << "Only support req=kWriteTo in pooling operations";
  int height = ishape[2], width = ishape[3];
  if (corner_pooling_type == 0 || corner_pooling_type == 1) { //top or bottom
    int h_end = 0,h_start = 0 , h_step = 0;
    if (corner_pooling_type == 0) {
        h_step = -1;
        h_start = height - 1;
        h_end = -1;
    } else {
        h_step = +1;
        h_start = 0;
        h_end = height;
    }
    const int count = ishape[0] * ishape[1] * width;
    const int gridSize = (count + mshadow::cuda::kMaxThreadsPerBlock - 1) / mshadow::cuda::kMaxThreadsPerBlock;
    dim3 dimGrid(mshadow::cuda::kMaxGridDim, (gridSize + mshadow::cuda::kMaxGridDim - 1) / mshadow::cuda::kMaxGridDim);
    dim3 dimBlock(mshadow::cuda::kMaxThreadsPerBlock);
    mshadow::cuda::CheckLaunchParam(dimGrid, dimBlock, "Corner Pooling Forward");
    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
    CornerPoolingForwardTBKernel<DType><<<  dimGrid, dimBlock, 0, stream>>>( count,
	in_data, ishape[0], ishape[1], ishape[2], ishape[3], out_data,
	h_step, h_start, h_end);
    MSHADOW_CUDA_POST_KERNEL_CHECK(CornerPoolingForwardTBKernel);
  } else if (corner_pooling_type == 2 || corner_pooling_type == 3) { //left or right
    int w_end = 0,w_start = 0 , w_step = 0;
    if (corner_pooling_type == 2) {
        w_step = -1;
        w_start = width - 1;
        w_end = -1;
    } else {
        w_step = +1;
        w_start = 0;
        w_end = width;
    }
    const int count = ishape[0] * ishape[1] * height;
    const int gridSize = (count + mshadow::cuda::kMaxThreadsPerBlock - 1) / mshadow::cuda::kMaxThreadsPerBlock;
    dim3 dimGrid(mshadow::cuda::kMaxGridDim, (gridSize + mshadow::cuda::kMaxGridDim - 1) / mshadow::cuda::kMaxGridDim);
    dim3 dimBlock(mshadow::cuda::kMaxThreadsPerBlock);
    mshadow::cuda::CheckLaunchParam(dimGrid, dimBlock, "Corner Pooling Forward");
    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
    CornerPoolingForwardLRKernel<DType><<< dimGrid, dimBlock, 0, stream>>>(count, 
	in_data, ishape[0], ishape[1], ishape[2], ishape[3], out_data,
	w_step, w_start, w_end);
    MSHADOW_CUDA_POST_KERNEL_CHECK(CornerPoolingForwardLRKernel);

  } else {
    LOG(FATAL) << "Unsupported corner pooling type";
  }

}

template<typename DType>
inline void corner_pool_grad(mshadow::Stream<gpu>* s, const DType* out_grad, const DType* in_data,
                   const DType* out_data, const TShape& ishape, 
                   const int corner_pooling_type, OpReqType req_type, DType* in_grad) { 
  const int height = ishape[2], width = ishape[3];
  if (corner_pooling_type == 0 || corner_pooling_type == 1) { //top or bottom
    int h_end = 0,h_start = 0 , h_step = 0;
    if (corner_pooling_type == 0) {
        h_step = -1;
        h_start = height - 1;
        h_end = -1;
    } else {
        h_step = +1;
        h_start = 0;
        h_end = height;
    }
    
    const int count = ishape[0] * ishape[1] * width;
    const int gridSize = (count + mshadow::cuda::kMaxThreadsPerBlock - 1) / mshadow::cuda::kMaxThreadsPerBlock;
    dim3 dimGrid(mshadow::cuda::kMaxGridDim, (gridSize + mshadow::cuda::kMaxGridDim - 1) / mshadow::cuda::kMaxGridDim);
    dim3 dimBlock(mshadow::cuda::kMaxThreadsPerBlock);
    mshadow::cuda::CheckLaunchParam(dimGrid, dimBlock, "Corner Pooling Backward");
    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
    CornerPoolingBackwardTBKernel<DType><<< dimGrid, dimBlock, 0, stream>>>(count, 
	out_data, ishape[0], ishape[1], ishape[2], ishape[3], out_grad, in_grad,
	h_step, h_start, h_end);
    MSHADOW_CUDA_POST_KERNEL_CHECK(CornerPoolingBackwardTBKernel);
    

  } else if (corner_pooling_type == 2 || corner_pooling_type == 3) { //left or right
    int w_end = 0,w_start = 0 , w_step = 0;
    if (corner_pooling_type == 2) {
        w_step = -1;
        w_start = width - 1;
        w_end = -1;
    } else {
        w_step = +1;
        w_start = 0;
        w_end = width;
    }
    const int count = ishape[0] * ishape[1] * height;
    const int gridSize = (count + mshadow::cuda::kMaxThreadsPerBlock - 1) / mshadow::cuda::kMaxThreadsPerBlock;
    dim3 dimGrid(mshadow::cuda::kMaxGridDim, (gridSize + mshadow::cuda::kMaxGridDim - 1) / mshadow::cuda::kMaxGridDim);
    dim3 dimBlock(mshadow::cuda::kMaxThreadsPerBlock);
    mshadow::cuda::CheckLaunchParam(dimGrid, dimBlock, "Corner Pooling Backward");
    hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
    CornerPoolingBackwardLRKernel<DType><<<dimGrid, dimBlock, 0, stream>>>(count, 
	out_data, ishape[0], ishape[1], ishape[2], ishape[3], out_grad, in_grad,
	w_step, w_start, w_end);
    MSHADOW_CUDA_POST_KERNEL_CHECK(CornerPoolingBackwardLRKernel);

  }
} 


NNVM_REGISTER_OP(CornerPooling)
.set_attr<FCompute>("FCompute<gpu>", CornerPoolingCompute<gpu>);

NNVM_REGISTER_OP(_backward_CornerPooling)
.set_attr<FCompute>("FCompute<gpu>", CornerPoolingGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet
